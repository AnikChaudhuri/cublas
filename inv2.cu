
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hipblas.h>



int main() {

const unsigned int N = 2;

const unsigned int Nmatrices = 2;

hipblasHandle_t handle;
hipblasCreate(&handle);

// --- Matrices to be inverted
float *h_A = new float[N*N*Nmatrices];
float *r_A = new float[N*N*Nmatrices];//result

h_A[0] = 4.f;
h_A[1] = 3.f;
h_A[2] = 8.f;
h_A[3] = 9.f;
h_A[4] = 5.f;
h_A[5] = 1.f;
h_A[6] = 2.f;
h_A[7] = 7.f;


// --- Allocate device matrices
float *d_A; hipMalloc((void**)&d_A, N*N*Nmatrices*sizeof(float));
float *c_A; hipMalloc((void**)&c_A, N*N*Nmatrices*sizeof(float));

// --- Move the matrix to be inverted from host to device
hipMemcpy(d_A,h_A,N*N*Nmatrices*sizeof(float),hipMemcpyHostToDevice);

// --- Creating the array of pointers needed as input to the batched getrf
float **h_inout_pointers = (float **)malloc(Nmatrices*sizeof(float *));
//for (int i=0; i<Nmatrices; i++) h_inout_pointers[i]=(float *)((char*)d_A+i*((size_t)N*N)*sizeof(float));
h_inout_pointers[0]=d_A;
h_inout_pointers[1]=d_A+N*N;

float **d_inout_pointers;
hipMalloc((void**)&d_inout_pointers, Nmatrices*sizeof(float *));
hipMemcpy(d_inout_pointers,h_inout_pointers,Nmatrices*sizeof(float *),hipMemcpyHostToDevice);
//free(h_inout_pointers);

float **r_inout_pointers = (float **)malloc(Nmatrices*sizeof(float *));
//for (int i=0; i<Nmatrices; i++) h_inout_pointers[i]=(float *)((char*)d_A+i*((size_t)N*N)*sizeof(float));
r_inout_pointers[0]=c_A;
r_inout_pointers[1]=c_A+N*N;

float **rd_inout_pointers;
hipMalloc((void**)&rd_inout_pointers, Nmatrices*sizeof(float *));
hipMemcpy(rd_inout_pointers,r_inout_pointers,Nmatrices*sizeof(float *),hipMemcpyHostToDevice);

int *d_PivotArray; hipMalloc((void**)&d_PivotArray, N*Nmatrices*sizeof(int));
int *d_InfoArray;  hipMalloc((void**)&d_InfoArray,  Nmatrices*sizeof(int));

int *h_PivotArray = (int *)malloc(N*Nmatrices*sizeof(int));
int *h_InfoArray  = (int *)malloc(  Nmatrices*sizeof(int));

hipblasSgetrfBatched(handle, N, d_inout_pointers, N, d_PivotArray, d_InfoArray, Nmatrices);
//cublasSafeCall(cublasSgetrfBatched(handle, N, d_inout_pointers, N, NULL, d_InfoArray, Nmatrices));

//gpuErrchk(cudaMemcpy(h_InfoArray,d_InfoArray,Nmatrices*sizeof(int),cudaMemcpyDeviceToHost));
hipblasSgetriBatched(handle, N, d_inout_pointers, N, d_PivotArray, rd_inout_pointers, N, d_InfoArray,
Nmatrices);


hipMemcpy(h_A,d_A,N*N*sizeof(float),hipMemcpyDeviceToHost);
hipMemcpy(r_A,c_A,Nmatrices*N*N*sizeof(float),hipMemcpyDeviceToHost);
//gpuErrchk(cudaMemcpy(h_PivotArray,d_PivotArray,N*Nmatrices*sizeof(int),cudaMemcpyDeviceToHost));

for (int i=0; i<N*N*Nmatrices; i++) printf("A[%i]=%f\n", i, r_A[i]);
hipFree(c_A);
hipFree(d_A);
hipFree(d_inout_pointers);
hipFree(rd_inout_pointers);
hipFree(d_InfoArray);
hipFree(d_PivotArray);
hipblasDestroy(handle);
return 0;
}