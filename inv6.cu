
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hipblas.h>

__global__ void copy(float** a, float* b, float** c, float* d, int count, float** e, float* f){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(idx < count){
        a[idx] = b + idx*2*2 ;
        c[idx] = d + idx*2*2;
        e[idx] = f + idx*2*2;
    }
    
}

int main() {

int N = 2;

int Nmatrices = 3;//number of batches

hipblasHandle_t handle;
hipblasCreate(&handle);

// --- Matrices to be inverted
float *h_A = new float[N*N*Nmatrices];
float *h_B = new float[N*N*Nmatrices];
float *r_A = new float[N*N*Nmatrices];//result

h_A[0] = 4.f;
h_A[1] = 3.f;
h_A[2] = 8.f;
h_A[3] = 9.f;
h_A[4] = 4.f;
h_A[5] = 3.f;
h_A[6] = 8.f;
h_A[7] = 9.f;
h_A[8] = 4.f;
h_A[9] = 3.f;
h_A[10] = 8.f;
h_A[11] = 9.f;

h_B[0] = 4.f;
h_B[1] = 3.f;
h_B[2] = 8.f;
h_B[3] = 9.f;
h_B[4] = 4.f;
h_B[5] = 3.f;
h_B[6] = 8.f;
h_B[7] = 9.f;
h_B[8] = 4.f;
h_B[9] = 3.f;
h_B[10] = 8.f;
h_B[11] = 9.f;


int count = Nmatrices;
// --- Allocate device matrices
float *d_A; hipMalloc((void**)&d_A, N*N*Nmatrices*sizeof(float));
float *d_B; hipMalloc((void**)&d_B, N*N*Nmatrices*sizeof(float));
float *c_A; hipMalloc((void**)&c_A, N*N*Nmatrices*sizeof(float));

hipMemcpy(d_A,h_A,N*N*Nmatrices*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(d_B,h_B,N*N*Nmatrices*sizeof(float),hipMemcpyHostToDevice);

float **d_inout_pointers;
hipMalloc((void**)&d_inout_pointers, Nmatrices*sizeof(float *));

float **db_inout_pointers;
hipMalloc((void**)&db_inout_pointers, Nmatrices*sizeof(float *));

float **rd_inout_pointers;
hipMalloc((void**)&rd_inout_pointers, Nmatrices*sizeof(float *));

    int lda, ldb, ldc;
    lda = ldb = ldc = 2;
    float alpha = 1; float beta = 0;
    copy<<<1,10>>>(d_inout_pointers, d_A, rd_inout_pointers, c_A, count, db_inout_pointers, d_B);
    
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_inout_pointers, lda, d_inout_pointers, ldb, &beta,rd_inout_pointers,ldc,Nmatrices);


    hipMemcpy(h_A,d_A,N*N*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(r_A,c_A,Nmatrices*N*N*sizeof(float),hipMemcpyDeviceToHost);
    //gpuErrchk(cudaMemcpy(h_PivotArray,d_PivotArray,N*Nmatrices*sizeof(int),cudaMemcpyDeviceToHost));

    for (int i=0; i<N*N*Nmatrices; i++) printf("A[%i]=%f\n", i, r_A[i]);
    
hipFree(c_A);
hipFree(d_A);
hipFree(d_B);
hipFree(d_inout_pointers);
hipFree(rd_inout_pointers);
hipFree(db_inout_pointers);
hipblasDestroy(handle);


return 0;
}