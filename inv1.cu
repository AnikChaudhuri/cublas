#include <hipblas.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>

int main(){
    int nr_rows_A, nr_cols_A;
    // for simplicity we are going to use square arrays
    nr_rows_A = nr_cols_A  = 2;
    int m, n, k, lda, ldb, ldc; 
    m = n = k = lda = ldb = ldc= nr_rows_A;
	int N = 2;
    
    float *a, *b, *c;
    float *d_A, *d_B, *d_C;

	a = (float*)malloc(N*N*sizeof(float));
	b = (float*)malloc(N*N*sizeof(float));
	c = (float*)malloc(N*N*sizeof(float));

	hipMalloc( (void**)&d_A, N * N * sizeof(float) );
    hipMalloc( (void**)&d_B, N * N * sizeof(float) );
    hipMalloc( (void**)&d_C, N * N * sizeof(float) );

   a[0] = 1; a[1] = 2; a[2] = 3; a[3] = 4;
   b[0] = 5; b[1] = 6; b[2] = 7; b[3] = 8;

	hipMemcpy(d_A, a, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, c, N*N*sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    float alpha = 1;
    float beta =0;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);

	hipMemcpy(a, d_A, N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_B, N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
        std::cout<<c[i]<<std::endl;
    }
    thrust::minstd_rand rng;
    // create a uniform_real_distribution to produce floats from [-7,13)
  thrust::uniform_real_distribution<float> dist(-7,13);
  std::cout << dist(rng) << std::endl;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipblasDestroy(handle);

}